#include "hip/hip_runtime.h"
#ifndef CUDA_NAIVE_KERNEL_BITWISE_CU
#define CUDA_NAIVE_KERNEL_BITWISE_CU

#include "../_shared/bitwise-cols/bitwise_ops_cuda_bit_ops.cuh"
#include "../_shared/bitwise-cols/bitwise_ops_macros.hpp"
#include "./models.hpp"
#include "gol_cuda_naive_bitwise.hpp"
#include <hip/hip_runtime.h>
#include "../../infrastructure/timer.hpp"

namespace algorithms {

using idx_t = std::int64_t;

__device__ inline idx_t get_idx(idx_t x, idx_t y, idx_t x_size) {
    return y * x_size + x;
}

template <typename col_type>
__device__ inline col_type load(idx_t x, idx_t y, BitGridOnCuda<col_type> data) {
    if (x < 0 || y < 0 || x >= data.x_size || y >= data.y_size)
        return 0;

    return data.input[get_idx(x, y, data.x_size)];
}

template <typename col_type>
__global__ void game_of_live_kernel(BitGridOnCuda<col_type> data) {
    idx_t x = blockIdx.x * blockDim.x + threadIdx.x;
    idx_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= data.x_size || y >= data.y_size)
        return;

    col_type lt = load(x - 1, y - 1, data);
    col_type ct = load(x, y - 1, data);
    col_type rt = load(x + 1, y - 1, data);

    col_type lc = load(x - 1, y, data);
    col_type cc = load(x, y, data);
    col_type rc = load(x + 1, y, data);

    col_type lb = load(x - 1, y + 1, data);
    col_type cb = load(x, y + 1, data);
    col_type rb = load(x + 1, y + 1, data);

    data.output[get_idx(x, y, data.x_size)] =
        CudaBitwiseOps<col_type>::compute_center_col(lt, ct, rt, lc, cc, rc, lb, cb, rb);
}

template <std::size_t Bits>
void GoLCudaNaiveBitwise<Bits>::run_kernel(size_type iterations) { // Added template parameter
    dim3 block(16, 16);
    dim3 grid((cuda_data.x_size + block.x - 1) / block.x, (cuda_data.y_size + block.y - 1) / block.y);

    infrastructure::StopWatch stop_watch(this->params.max_runtime_seconds);
    _performed_iterations = this->params.iterations;

    for (std::size_t i = 0; i < iterations; ++i) {
        if (stop_watch.time_is_up()) {
            _performed_iterations = i;
            return;
        }
        
        if (i != 0) {
            std::swap(cuda_data.input, cuda_data.output);
        }

        game_of_live_kernel<<<grid, block>>>(cuda_data);
    }
}

} // namespace algorithms

template class algorithms::GoLCudaNaiveBitwise<16>;
template class algorithms::GoLCudaNaiveBitwise<32>;
template class algorithms::GoLCudaNaiveBitwise<64>;

#endif // CUDA_NAIVE_KERNEL_CU