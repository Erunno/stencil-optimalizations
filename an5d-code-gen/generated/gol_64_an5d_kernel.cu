#include "hip/hip_runtime.h"
#include "gol_64_an5d_kernel.hu"
typedef ui64_t (*c_grid_t)[y_size_ext][x_size_ext];
__device__ unsigned long long __sbref_wrap(unsigned long long *sb, size_t index) { return sb[index]; }

__global__ void kernel0_4(unsigned long long *grid, int x_size_ext, int y_size_ext, int iters, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (iters - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (y_size_ext - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (x_size_ext - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __side0Len = 4;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 24;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    unsigned long long __reg_0;
    unsigned long long __reg_1_0;
    unsigned long long __reg_1_1;
    unsigned long long __reg_1_2;
    unsigned long long __reg_2_0;
    unsigned long long __reg_2_1;
    unsigned long long __reg_2_2;
    unsigned long long __reg_3_0;
    unsigned long long __reg_3_1;
    unsigned long long __reg_3_2;
    unsigned long long __reg_4_0;
    unsigned long long __reg_4_1;
    unsigned long long __reg_4_2;
    __shared__ unsigned long long __a_sb_double[__blockSize * 2];
    unsigned long long *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __writeValid4 = __updateValid && __local_c2 >= (__halo2 * 4) && __local_c2 < __side2LenOl - (__halo2 * 4);
    const AN5D_TYPE __storeValid = __writeValid4;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = grid[((__c0 % 2) * y_size_ext + __c1) * x_size_ext + __c2]; }} while (0)
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = GOL_OP((__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { unsigned long long etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { unsigned long long etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC2(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC3(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC4(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid4) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_0);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_0);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
      __STORE(1, __reg_4_1);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
      __STORE(2, __reg_4_2);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
      __STORE(3, __reg_4_0);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
      __STORE(4, __reg_4_1);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
      __LOAD(__reg_0, 7);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
      __LOAD(__reg_0, 8);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
      __STORE(4, __reg_4_1);
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 9; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 4, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h - 4, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
        __STORE(__h - 4, __reg_4_1);
        __h++;
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 4, __reg_4_2);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h - 3, __reg_4_0);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_2, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
        __STORE(__h - 2, __reg_4_1);
        __reg_3_0 = __reg_2_0;
        __CALC4(__reg_4_1, __reg_4_1, __reg_4_2, __reg_3_0);
        __STORE(__h - 1, __reg_4_2);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 4, __reg_4_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h - 3, __reg_4_0);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
        __STORE(__h - 2, __reg_4_1);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 1, __reg_4_2);
        __reg_3_1 = __reg_2_1;
        __CALC4(__reg_4_2, __reg_4_2, __reg_4_0, __reg_3_1);
        __STORE(__h + 0, __reg_4_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 4, __reg_4_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h - 3, __reg_4_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
        __STORE(__h - 2, __reg_4_1);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 1, __reg_4_2);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h + 0, __reg_4_0);
        __reg_3_2 = __reg_2_2;
        __CALC4(__reg_4_0, __reg_4_0, __reg_4_1, __reg_3_2);
        __STORE(__h + 1, __reg_4_1);
      }
    }
    else
    {
      for (__h = 9; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
        __STORE(__h - 4, __reg_4_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
        __STORE(__h - 4, __reg_4_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
        __STORE(__h - 4, __reg_4_1);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __CALC4(__reg_4_1, __reg_4_0, __reg_4_2, __reg_3_0);
      __STORE(__h - 4, __reg_4_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __CALC4(__reg_4_2, __reg_4_1, __reg_4_0, __reg_3_1);
      __STORE(__h - 4, __reg_4_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __CALC4(__reg_4_0, __reg_4_2, __reg_4_1, __reg_3_2);
      __STORE(__h - 4, __reg_4_1);
      __h++;
    }
}
__global__ void kernel0_3(unsigned long long *grid, int x_size_ext, int y_size_ext, int iters, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (iters - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (y_size_ext - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (x_size_ext - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __side0Len = 3;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 26;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    unsigned long long __reg_0;
    unsigned long long __reg_1_0;
    unsigned long long __reg_1_1;
    unsigned long long __reg_1_2;
    unsigned long long __reg_2_0;
    unsigned long long __reg_2_1;
    unsigned long long __reg_2_2;
    unsigned long long __reg_3_0;
    unsigned long long __reg_3_1;
    unsigned long long __reg_3_2;
    __shared__ unsigned long long __a_sb_double[__blockSize * 2];
    unsigned long long *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __writeValid3 = __updateValid && __local_c2 >= (__halo2 * 3) && __local_c2 < __side2LenOl - (__halo2 * 3);
    const AN5D_TYPE __storeValid = __writeValid3;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = grid[((__c0 % 2) * y_size_ext + __c1) * x_size_ext + __c2]; }} while (0)
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = GOL_OP((__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { unsigned long long etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { unsigned long long etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC2(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC3(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid3) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_0);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __STORE(1, __reg_3_1);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __STORE(2, __reg_3_2);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(3, __reg_3_0);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __LOAD(__reg_0, 5);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __LOAD(__reg_0, 6);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(3, __reg_3_0);
      __DB_SWITCH(); __syncthreads();
    }
    __a_sb = __a_sb_double + __blockSize * 0;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 7; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 3, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 3, __reg_3_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __reg_2_1 = __reg_1_1;
        __CALC3(__reg_3_2, __reg_3_2, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
        __reg_2_2 = __reg_1_2;
        __CALC3(__reg_3_0, __reg_3_0, __reg_3_1, __reg_2_2);
        __STORE(__h + 0, __reg_3_1);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 2, __reg_3_2);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 1, __reg_3_0);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h + 0, __reg_3_1);
        __reg_2_0 = __reg_1_0;
        __CALC3(__reg_3_1, __reg_3_1, __reg_3_2, __reg_2_0);
        __STORE(__h + 1, __reg_3_2);
      }
    }
    else
    {
      for (__h = 7; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
        __STORE(__h - 3, __reg_3_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
        __STORE(__h - 3, __reg_3_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
        __STORE(__h - 3, __reg_3_0);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __CALC3(__reg_3_0, __reg_3_2, __reg_3_1, __reg_2_2);
      __STORE(__h - 3, __reg_3_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __CALC3(__reg_3_1, __reg_3_0, __reg_3_2, __reg_2_0);
      __STORE(__h - 3, __reg_3_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __CALC3(__reg_3_2, __reg_3_1, __reg_3_0, __reg_2_1);
      __STORE(__h - 3, __reg_3_0);
      __h++;
    }
}
__global__ void kernel0_2(unsigned long long *grid, int x_size_ext, int y_size_ext, int iters, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (iters - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (y_size_ext - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (x_size_ext - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __side0Len = 2;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 28;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    unsigned long long __reg_0;
    unsigned long long __reg_1_0;
    unsigned long long __reg_1_1;
    unsigned long long __reg_1_2;
    unsigned long long __reg_2_0;
    unsigned long long __reg_2_1;
    unsigned long long __reg_2_2;
    __shared__ unsigned long long __a_sb_double[__blockSize * 2];
    unsigned long long *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __writeValid2 = __updateValid && __local_c2 >= (__halo2 * 2) && __local_c2 < __side2LenOl - (__halo2 * 2);
    const AN5D_TYPE __storeValid = __writeValid2;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = grid[((__c0 % 2) * y_size_ext + __c1) * x_size_ext + __c2]; }} while (0)
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = GOL_OP((__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { unsigned long long etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { unsigned long long etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __CALC2(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid2) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __STORE(1, __reg_2_1);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(2, __reg_2_2);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __LOAD(__reg_0, 3);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __LOAD(__reg_0, 4);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(2, __reg_2_2);
      __DB_SWITCH(); __syncthreads();
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 5; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 2, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h - 2, __reg_2_2);
        __h++;
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __reg_1_2 = __reg_0;
        __CALC2(__reg_2_0, __reg_2_0, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
        __reg_1_0 = __reg_0;
        __CALC2(__reg_2_1, __reg_2_1, __reg_2_2, __reg_1_0);
        __STORE(__h + 0, __reg_2_2);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 1, __reg_2_1);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h + 0, __reg_2_2);
        __reg_1_1 = __reg_0;
        __CALC2(__reg_2_2, __reg_2_2, __reg_2_0, __reg_1_1);
        __STORE(__h + 1, __reg_2_0);
      }
    }
    else
    {
      for (__h = 5; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
        __STORE(__h - 2, __reg_2_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
        __STORE(__h - 2, __reg_2_1);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
        __STORE(__h - 2, __reg_2_2);
        __h++;
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __CALC2(__reg_2_2, __reg_2_1, __reg_2_0, __reg_1_1);
      __STORE(__h - 2, __reg_2_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __CALC2(__reg_2_0, __reg_2_2, __reg_2_1, __reg_1_2);
      __STORE(__h - 2, __reg_2_1);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __CALC2(__reg_2_1, __reg_2_0, __reg_2_2, __reg_1_0);
      __STORE(__h - 2, __reg_2_2);
      __h++;
    }
}
__global__ void kernel0_1(unsigned long long *grid, int x_size_ext, int y_size_ext, int iters, int c0)
{
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
    const AN5D_TYPE __c0Len = (iters - 0);
    const AN5D_TYPE __c0Pad = (0);
    #define __c0 c0
    const AN5D_TYPE __c1Len = (y_size_ext - 1 - 1);
    const AN5D_TYPE __c1Pad = (1);
    #define __c1 c1
    const AN5D_TYPE __c2Len = (x_size_ext - 1 - 1);
    const AN5D_TYPE __c2Pad = (1);
    #define __c2 c2
    const AN5D_TYPE __halo1 = 1;
    const AN5D_TYPE __halo2 = 1;
    const AN5D_TYPE __side0Len = 1;
    const AN5D_TYPE __side1Len = 128;
    const AN5D_TYPE __side2Len = 30;
    const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
    const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
    const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
    const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
    const AN5D_TYPE __blockSize = 1 * __side2LenOl;
    const AN5D_TYPE __side1Num = (__c1Len + __side1Len - 1) / __side1Len;
    const AN5D_TYPE __side2Num = (__c2Len + __side2Len - 1) / __side2Len;
    const AN5D_TYPE __tid = threadIdx.y * blockDim.x + threadIdx.x;
    const AN5D_TYPE __local_c2 = __tid;
    const AN5D_TYPE __c1Id = blockIdx.x / __side2Num;
    const AN5D_TYPE __c2 = (blockIdx.x % __side2Num) * __side2Len + __local_c2 + __c2Pad - __OlLen2;
    unsigned long long __reg_0;
    unsigned long long __reg_1_0;
    unsigned long long __reg_1_1;
    unsigned long long __reg_1_2;
    __shared__ unsigned long long __a_sb_double[__blockSize * 2];
    unsigned long long *__a_sb = __a_sb_double;
    const AN5D_TYPE __loadValid = 1 && __c2 >= __c2Pad - __halo2 && __c2 < __c2Pad + __c2Len + __halo2;
    const AN5D_TYPE __updateValid = 1 && __c2 >= __c2Pad && __c2 < __c2Pad + __c2Len;
    const AN5D_TYPE __writeValid1 = __updateValid && __local_c2 >= (__halo2 * 1) && __local_c2 < __side2LenOl - (__halo2 * 1);
    const AN5D_TYPE __storeValid = __writeValid1;
    AN5D_TYPE __c1;
    AN5D_TYPE __h;
    const AN5D_TYPE __c1Pad2 = __c1Pad + __side1Len * __c1Id;
    #define __LOAD(reg, h) do { if (__loadValid) { __c1 = __c1Pad2 - __halo1 + h; reg = grid[((__c0 % 2) * y_size_ext + __c1) * x_size_ext + __c2]; }} while (0)
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_0_wrap(__rn0, __a) do { __rn0 = GOL_OP((__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_0(out, a) do { __CALCEXPR_0_wrap(out, a);  } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_1_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1)), (__pet_none), (__pet_none), (__pet_none)); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_1(out, a) do { unsigned long long etmp; __CALCEXPR_1_wrap(etmp, a); out += etmp; } while (0);
    #define __DEST (grid[(((c0 + 1) % 2) * y_size_ext + c1) * x_size_ext + c2])
    #define __REGREF(reg, i2) reg
    #define __SBREF(sb, i2) __sbref_wrap(sb, (int)__tid + i2)
    #define __CALCEXPR_2_wrap(__rn0, __a) do { __rn0 = GOL_OP((__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__pet_none), (__SBREF(__a_sb, -1)), (__REGREF(__a, 0)), (__SBREF(__a_sb, 1))); } while (0)
    #define __DB_SWITCH() do { __a_sb = &__a_sb_double[(__a_sb == __a_sb_double) ? __blockSize : 0]; } while (0)
    #define __CALCSETUP(a) do { __DB_SWITCH(); __a_sb[__tid] = a; __syncthreads(); } while (0)
    #define __CALCEXPR_2(out, a) do { unsigned long long etmp; __CALCEXPR_2_wrap(etmp, a); out += etmp; } while (0);
    #define __CALCEXPR(out0, out1, out2, reg) do { __CALCEXPR_0(out0, reg); __CALCEXPR_1(out1, reg); __CALCEXPR_2(out2, reg); } while (0);
    #define __CALC1(out0, out1, out2, reg) do { __CALCSETUP(reg); if (__writeValid1) { __CALCEXPR(out0, out1, out2, reg); } else out1 = reg; } while (0)
    #define __STORE(h, out) do { if (__storeValid) { __c1 = __c1Pad2 - __halo1 + h; __DEST = out; }} while (0)
    if (__c1Id == 0)
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(1, __reg_1_1);
    }
    else
    {
      __LOAD(__reg_0, 0);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __LOAD(__reg_0, 1);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __LOAD(__reg_0, 2);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(1, __reg_1_1);
    }
    __a_sb = __a_sb_double + __blockSize * 1;
    if (__c1Id == __side1Num - 1)
    {
      for (__h = 3; __h <= __c1Len - __side1Len * __c1Id + __halo1 * 2 - 4;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 1, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 1, __reg_1_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (0) {}
      else if (__h + 1 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_1, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
      }
      else if (__h + 2 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_2, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
      }
      else if (__h + 3 == __c1Len - __side1Len * __c1Id + __halo1 * 2)
      {
        __LOAD(__reg_0, __h + 0);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __LOAD(__reg_0, __h + 1);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h + 0, __reg_1_0);
        __LOAD(__reg_0, __h + 2);
        __CALC1(__reg_1_0, __reg_1_0, __reg_1_1, __reg_0);
        __STORE(__h + 1, __reg_1_1);
      }
    }
    else
    {
      for (__h = 3; __h <= __side1LenOl - 3;)
      {
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
        __STORE(__h - 1, __reg_1_2);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
        __STORE(__h - 1, __reg_1_0);
        __h++;
        __LOAD(__reg_0, __h);
        __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
        __STORE(__h - 1, __reg_1_1);
        __h++;
        __DB_SWITCH(); __syncthreads();
      }
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_1, __reg_1_0, __reg_1_2, __reg_0);
      __STORE(__h - 1, __reg_1_2);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_2, __reg_1_1, __reg_1_0, __reg_0);
      __STORE(__h - 1, __reg_1_0);
      __h++;
      if (__h == __side1LenOl) return;
      __LOAD(__reg_0, __h);
      __CALC1(__reg_1_0, __reg_1_2, __reg_1_1, __reg_0);
      __STORE(__h - 1, __reg_1_1);
      __h++;
    }
}
