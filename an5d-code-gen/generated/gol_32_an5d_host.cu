#include <assert.h>
#include <stdio.h>
#include "gol_32_an5d_kernel.hu"
#include <stdlib.h>
#include <stdint.h>
#include "cuda_gol_32_64_an5d.hpp"

#undef CELL_TYPE
#define CELL_TYPE ui32_t

CELL_TYPE GOL_OP(
    CELL_TYPE lt, CELL_TYPE tt, CELL_TYPE rt,
    CELL_TYPE lc, CELL_TYPE cc, CELL_TYPE rc,
    CELL_TYPE lb, CELL_TYPE cb, CELL_TYPE rb);

void cuda_gol_32(CELL_TYPE* src, CELL_TYPE* dst, int x_size, int y_size, int iters) {
    int x_size_ext = x_size + 2;
    int y_size_ext = y_size + 2;

    typedef CELL_TYPE (*c_grid_t)[y_size_ext][x_size_ext];

    int grid_size_bytes = sizeof(CELL_TYPE) * x_size_ext * y_size_ext;
    void* working_memory = malloc(2 * grid_size_bytes);

    for (int i = 0; i < 2 * grid_size_bytes; i++) {
        ((char*)working_memory)[i] = 0;
    }

    c_grid_t grid = (c_grid_t)working_memory;

    for (int y = 0; y < y_size; y++) {
        for (int x = 0; x < x_size; x++) {
            grid[0][y + 1][x + 1] = src[y * x_size + x];
        }
    }

    if (x_size_ext >= 3 && y_size_ext >= 3 && iters >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      unsigned int *dev_grid;
      
      cudaCheckReturn(hipMalloc((void **) &dev_grid, (size_t)(2) * (size_t)(y_size_ext) * (size_t)(x_size_ext) * sizeof(unsigned int)));
      
{
      cudaCheckReturn(hipMemcpy(dev_grid, grid, (size_t)(2) * (size_t)(y_size_ext) * (size_t)(x_size_ext) * sizeof(unsigned int), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (iters - 0);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (y_size_ext - 1 - 1);
      const AN5D_TYPE __c1Pad = (1);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (x_size_ext - 1 - 1);
      const AN5D_TYPE __c2Pad = (1);
      #define __c2 c2
      const AN5D_TYPE __halo1 = 1;
      const AN5D_TYPE __halo2 = 1;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 4;
        const AN5D_TYPE __side1Len = 128;
        const AN5D_TYPE __side2Len = 24;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          _32_kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 1)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 26;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
          }
        }
      }
      else if (__c0Len % __side0LenMax)
      {
        if (__c0Len % __side0LenMax == 1)
        {
          const AN5D_TYPE __side0Len = 1;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 30;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          _32_kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          const AN5D_TYPE __side0Len = 2;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 28;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          _32_kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          const AN5D_TYPE __side0Len = 3;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 26;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          _32_kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, x_size_ext, y_size_ext, iters, c0);
        }
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(grid, dev_grid, (size_t)(2) * (size_t)(y_size_ext) * (size_t)(x_size_ext) * sizeof(unsigned int), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_grid));
    }

    for (int y = 0; y < y_size; y++) {
        for (int x = 0; x < x_size; x++) {
            dst[y * x_size + x] = grid[iters % 2][y + 1][x + 1];
        }
    }

    free(working_memory);
}
